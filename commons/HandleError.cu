#include <stdio.h>
#include "colores.h"
#include "HandleError.h"


void HandleError(hipError_t err, const char *file, int line){
  if(err != hipSuccess){
    printf("%s in %s at line %d\n",hipGetErrorString(err),file,line);
    exit(EXIT_FAILURE);
  }
}

void CheckKernelSuccess(const char *file, int line){
  hipError_t err;
  err = hipGetLastError();
	if (err != hipSuccess){
	  sprintf(message,"Error: %s\n", hipGetErrorString(err));
		RED(message);
	}
}

