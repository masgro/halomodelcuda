#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include <assert.h>
#include <omp.h>
#include "constantes.h"
#include "colores.h"
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "HandleError.cu"

#ifdef DELTAS
#define ANCHO 0.01f
#define ALTURA_DELTA 2500.0f  //1/(2*ANCHO)/(2*ANCHO)
#endif

#define SIGMA    0.5f
#define SIGMA3 0.125f

#define BMAX 0.5f     
#define BMAX_YZ 0.3f  

#ifndef ANGULO
#define ANGULO 45
#endif

#define TRACERS_MASA_MIN 8.0f
#define TRACERS_MASA_MAX 16.0f

/*Cantidad total de hilos (RNG) que se van a tirar*/
#define RNGS 65536

/*Cantidad de veces que se lanza el Kernel de integracion*/
#define LAZOS 100
#define LAZOSPLUS 1000

/*Cantidad de Threads por Block*/
#ifndef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK 128
#endif

/*Tarjeta sobre la cual corre*/
#ifndef DEVICE
#define DEVICE 0
#endif

/*Intervalo espacial para la correlacion lineal*/
#define RANGO 10.0f

/*Numero de dimensiones de la integral*/
#ifdef DOSH
#define NDIM 14
#else
#define NDIM 5
#endif

/*Cantidad de pasos en cada direccion*/
#define NPASOS   100
#define PASOMIN -2.0
#define PASOMAX  2.0

/*Vectores integral y sigma, host version*/
float h_int[RNGS/THREADS_PER_BLOCK];
float h_sig[RNGS/THREADS_PER_BLOCK];

/*Vectores integral y sigma, device version*/
__device__ float d_int[RNGS/THREADS_PER_BLOCK];
__device__ float d_sig[RNGS/THREADS_PER_BLOCK];
__device__ float d_integral[1],d_sigma[1];

/*Coeficientes de la forma y normalizacion, host version*/
float h_bc[4][3];
float h_ab[4][3];
float h_norm[4];

/*Coeficientes de la forma y normalizacion, device version*/
__device__ float d_bc[4][3];
__device__ float d_ab[4][3];
__device__ float d_norm[4];

/*Coeficientes del alineamiento, host version*/
float h_alig[10][30][5];
float h_alig_norm[10][30];
float h_alig_m[11];
float h_alig_rmin;
float h_alig_rmax;
float h_alig_dr;

/*Coeficientes del alineamiento, device version*/
__device__ float d_alig[10][30][5];
__device__ float d_alig_norm[10][30];
__device__ float d_alig_m[11];
__device__ float d_alig_rmin;
__device__ float d_alig_rmax;
__device__ float d_alig_dr;

/*Vectores de limites de las integrales, host version*/
float h_xmin[NDIM];
float h_xmax[NDIM];
float norma_funmasa;

/*Vectores de limites de las integrales, device version*/
__constant__ float d_xmin[NDIM];
__constant__ float d_xmax[NDIM];

#include "lecturas.cu"

#include "chrono.c"

/*Incluye archivo con todas las funciones necesarias*/
#include "common_functions.cu"
#include "funciones.cu"

/*Kernel: toma un punto aleatorio en el espacio N-Dimensional
          y evalua la funcion integrando (T1h, T2h) en dicho punto.
          La evaluacion se guarda en d_int y el cuadrado en d_sig.
          Al final guarda el estado del RNG en state*/
__global__ void integra(hiprandState *state, float r, int eje){
  const unsigned int tid = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;

  /*Estado del RNG*/
  hiprandState xx = state[tid];

  const unsigned int it = threadIdx.x;
  int   i,j,k;
  float x[NDIM];
  float p[3];
  double value, sigma, tmp;

  __shared__ float xmin[NDIM], dx[NDIM];
  __shared__ float s_value[THREADS_PER_BLOCK];
  __shared__ float s_sigma[THREADS_PER_BLOCK];

  /*Inicializan variables*/
  if(it < NDIM){
    xmin[it] = d_xmin[it];
      dx[it] = d_xmax[it] - xmin[it];
  }

  /*Esperan hasta que todos terminen*/
  __syncthreads();

  /*Tira un numero random x de dimension NDIM*/
  value = 0.0; sigma = 0.0;
  for(j = 0; j < LAZOS; j++){
    k = 1;
    while(k != 0){
      x[NDIM-2] = dx[NDIM-2] * hiprand_uniform(&xx) + xmin[NDIM-2];
      x[NDIM-1] = dx[NDIM-1] * hiprand_uniform(&xx) + xmin[NDIM-1];

      if(eje == 2){ 
        p[2] = r*x[NDIM-2];
        p[1] = r*sqrtf(1.0f - x[NDIM-2]*x[NDIM-2])*sin(x[NDIM-1]);
        p[0] = r*sqrtf(1.0f - x[NDIM-2]*x[NDIM-2])*cos(x[NDIM-1]);
      }
      if(eje == 1){ 
        p[1] = r*x[NDIM-2];
        p[0] = r*sqrtf(1.0f - x[NDIM-2]*x[NDIM-2])*sin(x[NDIM-1]);
        p[2] = r*sqrt(1.0f - x[NDIM-2]*x[NDIM-2])*cos(x[NDIM-1]);
      }
      if(eje == 0){ 
        p[0] = r*x[NDIM-2];
        p[2] = r*sqrtf(1.0f - x[NDIM-2]*x[NDIM-2])*sin(x[NDIM-1]);
        p[1] = r*sqrtf(1.0f - x[NDIM-2]*x[NDIM-2])*cos(x[NDIM-1]);
      }

#ifdef DOSH
      for(i = 0; i < 6; i++)
        x[i] = dx[i] * hiprand_uniform(&xx) + xmin[i];
      
      x[6] = hiprand_normal(&xx);
      x[7] = hiprand_normal(&xx);
      x[8] = hiprand_normal(&xx);

      tmp = x[6]*x[6] + x[7]*x[7] + x[8]*x[8];
      if(tmp < 1.E-4){
        tmp = 1.0E-2/sqrt(tmp);
        x[6] *= tmp;
        x[7] *= tmp;
        x[8] *= tmp;
        tmp = 1.0E-4;
      }

      for(i = 9; i < NDIM-2; i++)
        x[i] = dx[i] * hiprand_uniform(&xx) + xmin[i];

      x[6] *= SIGMA;
      x[7] *= SIGMA;
      x[8] *= SIGMA;

      /*sqrt(2·pi)^3 sigma^3 / exp(-tmp/2)*/
      tmp  = SQRT_TWOPI_CUBO_CUDA*SIGMA3*exp(tmp*0.5f);
      tmp *= T2h(p,x);
#else 
      for(i = 0; i < NDIM-2; i++)
        x[i] = dx[i] * hiprand_uniform(&xx) + xmin[i];

      tmp = T1h(p,x);
#endif
      k = (isnan(tmp) + isinf(tmp));
    }

    value += tmp;
    sigma += tmp*tmp;
  }

  /*Guarda en la memoria compartida*/
  s_value[it] = value;
  s_sigma[it] = sigma;

  __syncthreads();

  if(it == 0){
    value = 0.0f; sigma = 0.0f;
    for(i = 0; i < THREADS_PER_BLOCK; i++){
      value += s_value[i];
      sigma += s_sigma[i];
    }
    j = blockIdx.x;
    /*Suma a la memoria global*/
    d_int[j] = value;
    d_sig[j] = sigma;
  }

  /*Guarda el estado del RNG*/
  state[tid] = xx;
}

__global__ void suma(void){
  const unsigned int tid = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
  int inext;

  inext = RNGS/THREADS_PER_BLOCK/2;
  while(inext >= 1){
    if(tid < inext){
      d_int[tid] += d_int[tid+inext];
      d_sig[tid] += d_sig[tid+inext];
    }
    inext = inext/2;
    __syncthreads();
  }

  if(tid == 0){
    d_integral[0] = d_int[0];
    d_sigma[0] = d_sig[0];
  }
}

/*Funcion que imprime las propiedades de la placa*/
__inline__ void printDevProp(hipDeviceProp_t devProp){
  printf("#############################################\n");
  printf(" Running on device:         %s\n",  devProp.name);
  printf(" Total global memory:       %zu\n", devProp.totalGlobalMem);
  printf(" Total shared memory/block: %zu\n", devProp.sharedMemPerBlock);
  printf(" Total registers/block:     %d\n",  devProp.regsPerBlock);
  printf(" Maximum threads/block:     %d\n",  devProp.maxThreadsPerBlock);
  printf(" Total constant memory:     %zu\n", devProp.totalConstMem);
  printf(" Kernel execution timeout:  %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
  printf(" WarpSize:                  %d\n",devProp.warpSize);
  printf(" Compute Capability:        %d%d\n",devProp.major,devProp.minor);
  printf("#############################################\n");
}

int setseed(void){
  int seed;
  FILE *pf;
  pf = fopen("/dev/urandom","r");
  fread(&seed,sizeof(int),1,pf);
  fclose(pf);
  return seed;
}

int main(int argc, char **argv){
  double time,elapsed;
  FILE  *pfout;
  char  filename[200],term[200];
  int   i,j,l;
  int   run;
  float r,s;
  float volumen;
  float h_radio;
  //float *d_radio;
  //int   *d_eje;
  hiprandState *devStates;

  elapsed = 0.0f;
  chrono(START,&time);

  run = (argc > 1)? atoi(argv[1]) : 0;

#ifdef DOSH
  sprintf(term,"2h");
#else
  sprintf(term,"1h");
#endif

  /*Setea el device a utilizar*/
  hipSetDevice(DEVICE);

  /*Lee e imprime las propiedades del device*/
  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, DEVICE);
  printDevProp(devProp);

  //HANDLE_ERROR(hipMalloc((void **)&d_radio,sizeof(float)));
  //HANDLE_ERROR(hipMalloc((void **)&d_eje,sizeof(int)));
  
  /*Chequea Cantidad de Threads y de Blocks*/
  assert(THREADS_PER_BLOCK <= 1024);
  assert(RNGS%THREADS_PER_BLOCK == 0); // should be divisible by blocks

  /*Setea Cantidad de Threads y de Blocks*/
  dim3 dimBlock(THREADS_PER_BLOCK,1,1);
  dim3 dimGrid(RNGS/THREADS_PER_BLOCK,1,1);

  fprintf(stdout,"Corriendo %d Blocks con %d threads cada uno\n",
                                RNGS/THREADS_PER_BLOCK,THREADS_PER_BLOCK);

  /*Allocatea memoria para el RNG*/
  HANDLE_ERROR(hipMalloc((void **)&devStates,RNGS*sizeof(hiprandState)));

  /*Setea la semilla*/
  int seed = setseed();

  /*Setea las semillas de los RNG en el device*/
  setup_kernel<<<dimGrid,dimBlock>>>(devStates,seed);
  hipDeviceSynchronize();

  /*lee los coeficientes de los ajustes*/
  read_coefficients();

  float Numin   = Nu_M(CENTROS_MASA_MIN);
  float Numax   = Nu_M(CENTROS_MASA_MAX);
  float ncmedio = nc_medio(Numin,Numax,dimGrid,dimBlock,devStates);
  //float ncmedio = nc_medio(CENTROS_MASA_MIN,CENTROS_MASA_MAX,dimGrid,dimBlock,devStates);

#ifdef CG 
  Numin   = Nu_M(TRACERS_MASA_MIN);
  Numax   = Nu_M(TRACERS_MASA_MAX);
  //float ngmedio = ng_medio(TRACERS_MASA_MIN,TRACERS_MASA_MAX,dimGrid,dimBlock,devStates);
  float ngmedio = ng_medio(Numin,Numax,dimGrid,dimBlock,devStates);
#endif

  /*Chequea si la integral del perfil hasta el radio virial da 1*/
  //normalizacion_perfil(dimGrid,dimBlock,devStates);
  
  prueba_ng_medio();

  /*Verifica que la integral de bias(nu)*f(nu) de 1*/
  prueba_bias_f(dimGrid,dimBlock,devStates);

  /*Calcula la normalizacion de la funcion de masa*/
  normalizacion_func_masa(dimGrid,dimBlock,devStates);

  /*Setea los limites de integracion*/
  /*Halo Centro*/
  h_xmin[0] = (float)CENTROS_MASA_MIN; /*Masa minima*/
  h_xmax[0] = (float)CENTROS_MASA_MAX; /*Masa maxima*/
  /*Forma*/
  h_xmin[1] = 0.10f; /* ab minimo */
  h_xmax[1] = 1.00f; /* ab maximo */
  h_xmin[2] = 0.10f; /* bc minimo */
  h_xmax[2] = 1.00f; /* bc maximo */

#ifdef DELTAS
  h_xmin[1] = ABMEDIO - ANCHO; /* ab minimo */
  h_xmax[1] = ABMEDIO + ANCHO; /* ab maximo */
  h_xmin[2] = BCMEDIO - ANCHO; /* bc minimo */
  h_xmax[2] = BCMEDIO + ANCHO; /* bc maximo */
#endif

#ifdef DOSH
  /*Halo Vecino*/
  h_xmin[3] = (float)TRACERS_MASA_MIN; /*Masa minima*/
  h_xmax[3] = (float)TRACERS_MASA_MAX; /*Masa maxima*/
  h_xmin[4] = 0.10f; /* ab minimo */
  h_xmax[4] = 1.00f; /* ab maximo */
  h_xmin[5] = 0.10f; /* bc minimo */
  h_xmax[5] = 1.00f; /* bc maximo */

#ifdef DELTAS
  h_xmin[4] = ABMEDIO - ANCHO; /* ab minimo */
  h_xmax[4] = ABMEDIO + ANCHO; /* ab maximo */
  h_xmin[5] = BCMEDIO - ANCHO; /* bc minimo */
  h_xmax[5] = BCMEDIO + ANCHO; /* bc maximo */
#endif

  /*Volumen*/
  //h_xmin[6] = -RANGO; 
  //h_xmax[6] = +RANGO;
  //h_xmin[7] = -RANGO;
  //h_xmax[7] = +RANGO;
  //h_xmin[8] = -RANGO;
  //h_xmax[8] = +RANGO;

  /*Orientacion del halo vecino*/
  h_xmin[9]  =  0.0f;
  h_xmax[9]  =  M_PI;
  h_xmin[10] =  0.0f;
  h_xmax[10] =  1.0f;
  h_xmin[11] =  0.0f;
  h_xmax[11] =  M_PI;
#endif

  h_xmin[NDIM-2] = cos((float)ANGULO*M_PI/180.0);
  h_xmax[NDIM-2] = 1.0f;
  h_xmin[NDIM-1] = 0.0f;
  h_xmax[NDIM-1] = 2.0f*M_PI;

#ifdef MERCHAN
  float norma_merchan = integra_merchan(dimGrid,dimBlock,devStates);
#endif
#ifdef DOSH
  float norma_align = integra_align(dimGrid,dimBlock,devStates);
#endif

  /*Copia los limites de integracion al device*/
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_xmin),h_xmin,NDIM*sizeof(float)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_xmax),h_xmax,NDIM*sizeof(float)));

  /*Calcula el hipervolumen de integracion*/
  volumen = 1.0f;
#ifdef DOSH
  /* PARA UTILIZAR RANDOM GAUSSIANO EN 6,7,8 */
  for(i = 0; i < 6 ; i++){
    printf("---- %d %f\n",i,(h_xmax[i] - h_xmin[i]));
    volumen *= (h_xmax[i] - h_xmin[i]);
  }
  for(i = 9; i < NDIM-2; i++){
    printf("---- %d %f\n",i,(h_xmax[i] - h_xmin[i]));
    volumen *= (h_xmax[i] - h_xmin[i]);
  }
#else
  for(i = 0; i < NDIM-2; i++){
    printf("---- %d %f\n",i,(h_xmax[i] - h_xmin[i]));
    volumen *= (h_xmax[i] - h_xmin[i]);
  }
#endif

  /*Calcula la memoria total en el device*/
  size_t memfree, memtot;
  HANDLE_ERROR(hipMemGetInfo(&memfree,&memtot));
  printf("Memoria CUDA Total: %8.3lf Mb Used: %8.3lf Mb free: %8.3lf Mb \n",
         (float)memtot/1024.0f/1024.0f,(float)(memtot-memfree)/1024.0f/1024.0f,
         (float)memfree/1024.0f/1024.0f);
  
  /*Imprime alguna informacion*/
  printf("--------------------------\n");
  printf("  Volumen:  %E\n",volumen);
  printf("  RHOMEDIO: %E\n",RHOMEDIO);
  printf("  NCMEDIO:  %E\n",ncmedio);
#ifdef CG
  printf("  NGMEDIO: %E\n",ngmedio);
#endif
#ifdef DOSH
  printf("  NormaAlign: %E\n",norma_align);
#endif
  printf("  RNGs:  %d\n",RNGS);
  printf("  LAZOS: %d\n",LAZOS);
  printf("--------------------------\n");

  printf("Computando termino de %s....\n",term);

  float dpaso;
  dpaso = (PASOMAX - PASOMIN)/(float)NPASOS;

  /*Recorre las 3 direcciones j=0(x),1(y),2(z)*/
  for(j = 0; j < 3; j++){
    /*Abre archivo de salida*/
    sprintf(filename,"funcorr_%1d_%s.%02d",j,term,run);
    pfout = fopen(filename,"w");

    /*En cada direccion hace NPASOS pasos*/
    for(i = 0; i < NPASOS; i++){
      /*Setea posicion en la direccion dada*/
      h_radio = dpaso*(float)(i) + PASOMIN;
      h_radio = powf(10.0f,h_radio);

      /*Copia posicion al device*/
      //HANDLE_ERROR(hipMemcpy(d_radio,&h_radio,sizeof(float),hipMemcpyHostToDevice));
      //HANDLE_ERROR(hipMemcpy(d_eje,&j,sizeof(int),hipMemcpyHostToDevice));

      /*Lanza kernel*/
      integra<<<dimGrid,dimBlock>>>(devStates,h_radio,j);
      hipDeviceSynchronize();

      //suma<<<dimGrid,dimBlock>>>();
      //hipDeviceSynchronize();
      /*Termina kernel*/

      CHECK_KERNEL_SUCCESS();

      ////*Copia sumatorias al host y termina de reducir en el host*/
      HANDLE_ERROR(hipMemcpyFromSymbol(h_int,HIP_SYMBOL(d_int),(RNGS/THREADS_PER_BLOCK)*sizeof(float)));
      HANDLE_ERROR(hipMemcpyFromSymbol(h_sig,HIP_SYMBOL(d_sig),(RNGS/THREADS_PER_BLOCK)*sizeof(float)));

      /*Termina de reducir en el host*/
      r = 0.0; s = 0.0;
      for(l = 0; l < RNGS/THREADS_PER_BLOCK; l++){
        if(isnan(h_int[l]))printf("%f %d %d\n",h_int[l],i,j);
        r += h_int[l];
        s += h_sig[l];
      }

      //HANDLE_ERROR(hipMemcpyFromSymbol(&r,HIP_SYMBOL(d_integral),sizeof(float)));
      //HANDLE_ERROR(hipMemcpyFromSymbol(&s,HIP_SYMBOL(d_sigma),sizeof(float)));

      /*Estima la integral y el sigma*/
      r /= (float)((long)RNGS*(long)LAZOS);
      s /= (float)((long)RNGS*(long)LAZOS);
      s -= (r*r);
      s /= (float)((long)RNGS*(long)LAZOS);
      s  = sqrt(s);
      s *= volumen;
      r *= volumen;

      r /= ncmedio;
      s /= ncmedio;

      r /= norma_merchan;
      s /= norma_merchan;

#ifdef DOSH
      r /= norma_align;
      s /= norma_align;
      r /= norma_merchan;
      s /= norma_merchan;
#endif

#ifdef CG
			r /= ngmedio;
			s /= ngmedio;
#endif
      /*Imprime en file de salida*/
      fprintf(pfout,"%e %e %e\n",h_radio,r,s);
    }
    /*Cierra archivo de salida*/
    fclose(pfout);
  }

  /*Libera memoria allocateada en el device*/
  HANDLE_ERROR(hipFree(devStates));

  /*Computa el tiempo total utilizado en el device*/
  chrono(STOP,&time);
  elapsed += time;
  printf("Tiempo: %lf [seg] \n", elapsed);

  /*Fin del programa*/
  return(EXIT_SUCCESS);
}
