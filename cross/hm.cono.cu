#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include <assert.h>
#include <omp.h>
#include "constantes.h"
#include "colores.h"
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "HandleError.cu"

#define SIGMA   0.50000f
#define SIGMA2  0.25000f
#define SIGMA3  0.12500f

#ifndef ANGULO
#define ANGULO 45
#endif

#define TRACERS_MASA_MIN 9.0f
#define TRACERS_MASA_MAX 15.5f

/*Cantidad total de hilos (RNG) que se van a tirar*/
#define RNGS 65536

/*Cantidad de veces que se lanza el Kernel de integracion*/
#ifdef DOSH
#define LAZOS 1000
#else
#define LAZOS 100
#endif
#define LAZOSPLUS 1000

/*Cantidad de Threads por Block*/
#ifndef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK 128
#endif

/*Tarjeta sobre la cual corre*/
#ifndef DEVICE
#define DEVICE 0
#endif

/*Intervalo espacial para la correlacion lineal*/
#define RANGO 10.0f

/*Numero de dimensiones de la integral*/
#ifdef DOSH
#define NDIM 14
#else
#define NDIM 5
#endif

/*Cantidad de pasos en cada direccion*/
//#define NPASOS   100
//#define PASOMIN -1.00
//#define PASOMAX  1.85

//#ifndef DOSH
//#undef NPASOS
//#define NPASOS   20
//float h_radio_vector[NPASOS] = {0.119379,0.165646,0.229845,0.318927,0.442533,0.614046,0.852031,1.182253,
//                                1.640459,2.276252,3.158458,4.382582,6.081139,8.438007,11.708325,16.246120,
//                                22.542629,31.279476,43.402466,60.223957};

#define NPASOS   25
float h_radio_vector[NPASOS] = {1.1399923e-01,1.4815143e-01,1.9253504e-01,2.5021520e-01,3.2517537e-01,4.2259231e-01,5.4919362e-01,7.1372253e-01,9.2754149e-01,1.2054168e+00,
1.5665389e+00,2.0358467e+00,2.6457512e+00,3.4383726e+00,4.4684496e+00,5.8071198e+00,7.5468326e+00,9.8077326e+00,1.2745960e+01,1.6564426e+01,
2.1526846e+01,2.7975912e+01,3.6357018e+01,4.7248943e+01,6.1403919e+01};
//#endif

/*Vectores integral y sigma, host version*/
float h_int[RNGS/THREADS_PER_BLOCK];
float h_sig[RNGS/THREADS_PER_BLOCK];

/*Vectores integral y sigma, device version*/
__device__ float d_int[RNGS/THREADS_PER_BLOCK];
__device__ float d_sig[RNGS/THREADS_PER_BLOCK];
__device__ float d_integral[1],d_sigma[1];

/*Parametros forma y alineacion*/
float h_abmedio, h_bcmedio;
float h_align_b, h_align_c;
__device__ float d_abmedio, d_bcmedio;
__device__ float d_align_b, d_align_c;

/*Coeficientes de la forma y normalizacion, host version*/
float h_bc[4][3];
float h_ab[4][3];
float h_norm[4];

/*Coeficientes de la forma y normalizacion, device version*/
__device__ float d_bc[4][3];
__device__ float d_ab[4][3];
__device__ float d_norm[4];

/*Coeficientes del alineamiento, host version*/
float h_alig[10][30][5];
float h_alig_norm[10][30];
float h_alig_m[11];
float h_alig_rmin;
float h_alig_rmax;
float h_alig_dr;
float h_centros_masa_min;
float h_centros_masa_max;

/*Coeficientes del alineamiento, device version*/
__device__ float d_alig[10][30][5];
__device__ float d_alig_norm[10][30];
__device__ float d_alig_m[11];
__device__ float d_alig_rmin;
__device__ float d_alig_rmax;
__device__ float d_alig_dr;
__device__ float d_centros_masa_min;
__device__ float d_centros_masa_max;

/*Vectores de limites de las integrales, host version*/
float h_xmin[NDIM];
float h_xmax[NDIM];
float norma_funmasa;

/*Vectores de limites de las integrales, device version*/
__constant__ float d_xmin[NDIM];
__constant__ float d_xmax[NDIM];

#include "lecturas.cu"

#include "chrono.c"

/*Incluye archivo con todas las funciones necesarias*/
#include "common_functions.cu"
#include "funciones.cu"

  /*Kernel: toma un punto aleatorio en el espacio N-Dimensional
  y evalua la funcion integrando (T1h, T2h) en dicho punto.
  La evaluacion se guarda en d_int y el cuadrado en d_sig.
  Al final guarda el estado del RNG en state*/
__global__ void integra(hiprandState *state, float r, int eje){
  const unsigned int tid = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;

  /*Estado del RNG*/
  hiprandState seed = state[tid];

  const unsigned int it = threadIdx.x;
  int   i,j;
  float x[NDIM];
  float p[3];
  double value, sigma, tmp;

  __shared__ float xmin[NDIM], dx[NDIM];
  __shared__ float s_value[THREADS_PER_BLOCK];
  __shared__ float s_sigma[THREADS_PER_BLOCK];

  /*Inicializan variables*/
  if(it < NDIM){
    xmin[it] = d_xmin[it];
      dx[it] = d_xmax[it] - xmin[it];
  }

  /*Esperan hasta que todos terminen*/
  __syncthreads();

  value = 0.0; sigma = 0.0;
  for(j = 0; j < LAZOS; j++){
    /*Tira un numero random x de dimension NDIM*/
    do{
      x[NDIM-2] = dx[NDIM-2] * hiprand_uniform(&seed) + xmin[NDIM-2]; /*cos(theta)*/
      x[NDIM-1] = dx[NDIM-1] * hiprand_uniform(&seed) + xmin[NDIM-1]; /*phi*/

      if(eje == 2){ 
        p[2] = r*x[NDIM-2];
        p[1] = r*sqrtf(1.0f - x[NDIM-2]*x[NDIM-2])*sin(x[NDIM-1]);
        p[0] = r*sqrtf(1.0f - x[NDIM-2]*x[NDIM-2])*cos(x[NDIM-1]);
      }
      if(eje == 1){ 
        p[1] = r*x[NDIM-2];
        p[0] = r*sqrtf(1.0f - x[NDIM-2]*x[NDIM-2])*sin(x[NDIM-1]);
        p[2] = r*sqrt(1.0f - x[NDIM-2]*x[NDIM-2])*cos(x[NDIM-1]);
      }
      if(eje == 0){ 
        p[0] = r*x[NDIM-2];
        p[2] = r*sqrtf(1.0f - x[NDIM-2]*x[NDIM-2])*sin(x[NDIM-1]);
        p[1] = r*sqrtf(1.0f - x[NDIM-2]*x[NDIM-2])*cos(x[NDIM-1]);
      }


#ifdef DOSH
      for(i = 0; i < 6; i++)
        x[i] = dx[i] * hiprand_uniform(&seed) + xmin[i];
      
      /*Posicion del halo vecino respecto del punto p*/
      x[6] = hiprand_normal(&seed);
      x[7] = hiprand_normal(&seed);
      x[8] = hiprand_normal(&seed);

      tmp = x[6]*x[6] + x[7]*x[7] + x[8]*x[8];
      if(tmp < 1.E-4){ 
        /*Esto es para evitar que el halo vecino caiga muy cerca
        del punto p, si cae muy cerca la densidad revienta*/
        tmp = 1.0E-2/sqrt(tmp);
        x[6] *= tmp;
        x[7] *= tmp;
        x[8] *= tmp;
        tmp = 1.0E-4;
      }

      for(i = 9; i < NDIM-2; i++)
        x[i] = dx[i] * hiprand_uniform(&seed) + xmin[i];

      x[6] *= SIGMA;
      x[7] *= SIGMA;
      x[8] *= SIGMA;

      /*sqrt(2·pi)^3 sigma^3 / exp(-tmp/2)*/
      tmp  = SQRT_TWOPI_CUBO_CUDA*SIGMA3*exp(tmp*0.5f);
      tmp *= T2h(p,x);
#else
      for(i = 0; i < NDIM-2; i++)
        x[i] = dx[i] * hiprand_uniform(&seed) + xmin[i];

      tmp = T1h(p,x);
#endif
      if(isfinite(tmp))break;
    }while(1);

    value += tmp;
    sigma += tmp*tmp;
  }

  /*Guarda en la memoria compartida*/
  s_value[it] = value;
  s_sigma[it] = sigma;

  /*Esperan hasta que todos terminen*/
  __syncthreads();

  /*Thread 0 colecta los resultados del bloque y guarda en la memoria global*/
  if(it == 0){
    value = 0.0f; sigma = 0.0f;
    for(i = 0; i < THREADS_PER_BLOCK; i++){
      value += s_value[i];
      sigma += s_sigma[i];
    }
    j = blockIdx.x;

    /*Guarda en la memoria global*/
    d_int[j] = value;
    d_sig[j] = sigma;
  }

  /*Guarda el estado del RNG*/
  state[tid] = seed;
}

__global__ void suma(void){
  const unsigned int tid = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
  int inext;

  inext = RNGS/THREADS_PER_BLOCK/2;
  while(inext >= 1){
    if(tid < inext){
      d_int[tid] += d_int[tid+inext];
      d_sig[tid] += d_sig[tid+inext];
    }
    inext = inext/2;
    __syncthreads();
  }

  if(tid == 0){
    d_integral[0] = d_int[0];
    d_sigma[0] = d_sig[0];
  }
}

/*Funcion que imprime las propiedades de la placa*/
__inline__ void printDevProp(hipDeviceProp_t devProp){
  printf("#############################################\n");
  printf(" Running on device:         %s\n",  devProp.name);
  printf(" Total global memory:       %zu\n", devProp.totalGlobalMem);
  printf(" Total shared memory/block: %zu\n", devProp.sharedMemPerBlock);
  printf(" Total registers/block:     %d\n",  devProp.regsPerBlock);
  printf(" Maximum threads/block:     %d\n",  devProp.maxThreadsPerBlock);
  printf(" Total constant memory:     %zu\n", devProp.totalConstMem);
  printf(" Kernel execution timeout:  %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
  printf(" WarpSize:                  %d\n",devProp.warpSize);
  printf(" Compute Capability:        %d.%d\n",devProp.major,devProp.minor);
  printf("#############################################\n");
}

unsigned long long setseed(void){
  int s;
  unsigned long long seed;
  FILE *pf;
  pf = fopen("/dev/urandom","r");
  fread(&s,sizeof(int),1,pf);
  fclose(pf);
  seed = (unsigned long long)s;
  return(seed);
}

void set_variables(int argc, char **argv){

  if(argc < 5){
    printf("Usage: %s ABMEDIO BCMEDIO ALGN_C ALGN_B\n",argv[0]);
    exit(EXIT_FAILURE);
  }

  h_centros_masa_min = atof(argv[1]);
  h_centros_masa_max = atof(argv[2]);
  h_abmedio = atof(argv[3]);
  h_bcmedio = atof(argv[4]);
  h_align_b = atof(argv[5]);
  h_align_c = atof(argv[6]);

  printf("%.02f %.02f %.02f %.02f\n",h_abmedio,h_bcmedio,h_align_b,h_align_c);

  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_centros_masa_min),&h_centros_masa_min,sizeof(float)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_centros_masa_max),&h_centros_masa_max,sizeof(float)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_abmedio),&h_abmedio,sizeof(float)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_bcmedio),&h_bcmedio,sizeof(float)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_align_b),&h_align_b,sizeof(float)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_align_c),&h_align_c,sizeof(float)));
}

int main(int argc, char **argv){
  double time,elapsed;
  FILE  *pfout;
  char  filename[200],term[200];
  int   i,j,l;
  int   run;
  float r,s;
  float volumen;
  float h_radio;
  hiprandState *devStates;

  /*Setea el device a utilizar*/
  //hipSetDevice(DEVICE);
  hipGetDevice(&i);

  /*Lee e imprime las propiedades del device*/
  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, i);
  printDevProp(devProp);

  set_variables(argc,argv);

  elapsed = 0.0f;
  chrono(START,&time);

  run = (argc > 7)? atoi(argv[7]) : 0;

#ifdef DOSH
  sprintf(term,"2h");
#else
  sprintf(term,"1h");
#endif

  /*Chequea Cantidad de Threads y de Blocks*/
  assert(THREADS_PER_BLOCK <= 1024);
  assert(RNGS%THREADS_PER_BLOCK == 0); // should be divisible by blocks

  /*Setea Cantidad de Threads y de Blocks*/
  dim3 dimBlock(THREADS_PER_BLOCK,1,1);
  dim3 dimGrid(RNGS/THREADS_PER_BLOCK,1,1);

  fprintf(stdout,"Corriendo %d Blocks con %d threads cada uno\n",
                        RNGS/THREADS_PER_BLOCK,THREADS_PER_BLOCK);

  /*Allocatea memoria para el RNG*/
  HANDLE_ERROR(hipMalloc((void **)&devStates,RNGS*sizeof(hiprandState)));

  /*Setea la semilla*/
  unsigned long long seed = setseed();
  fprintf(stdout,"Seed %d\n",seed);

  /*Setea las semillas de los RNG en el device*/
  setup_kernel<<<dimGrid,dimBlock>>>(devStates,seed);
  hipDeviceSynchronize();

  /*lee los coeficientes de los ajustes*/
  read_coefficients();

  float Numin = Nu_M(h_centros_masa_min);
  float Numax = Nu_M(h_centros_masa_max);
  float ncmedio = nc_medio(Numin,Numax,dimGrid,dimBlock,devStates);
  /*FUNCION DE MASA FIT*/
  //float ncmedio = nc_medio(CENTROS_MASA_MIN,CENTROS_MASA_MAX,dimGrid,dimBlock,devStates);

#ifdef CG 
  Numin = Nu_M(TRACERS_MASA_MIN);
  Numax = Nu_M(TRACERS_MASA_MAX);
  float ngmedio = ng_medio(Numin,Numax,dimGrid,dimBlock,devStates);
  /*FUNCION DE MASA FIT*/
  //float ngmedio = ng_medio(TRACERS_MASA_MIN,TRACERS_MASA_MAX,dimGrid,dimBlock,devStates);
#endif

  /*Chequea si la integral del perfil hasta el radio virial da 1*/
#ifndef DOSH
  //float norma_perfil = normalizacion_perfil(dimGrid,dimBlock,devStates);
#endif
  
  //prueba_ng_medio();

  /*Verifica que la integral de bias(nu)*f(nu) de 1*/
  //prueba_bias_f(dimGrid,dimBlock,devStates);

  /*Calcula la normalizacion de la funcion de masa*/
  //normalizacion_func_masa(dimGrid,dimBlock,devStates);

  /*Setea los limites de integracion*/
  /*Halo Centro*/
  h_xmin[0] = (float)h_centros_masa_min; /*Masa minima*/
  h_xmax[0] = (float)h_centros_masa_max; /*Masa maxima*/
  /*Forma*/
  h_xmin[1] = 0.00f; /* ab minimo */
  h_xmax[1] = 1.00f; /* ab maximo */
  h_xmin[2] = 0.00f; /* bc minimo */
  h_xmax[2] = 1.00f; /* bc maximo */

#ifdef DOSH
  /*Halo Vecino*/
  h_xmin[3] = (float)TRACERS_MASA_MIN; /*Masa minima*/
  h_xmax[3] = (float)TRACERS_MASA_MAX; /*Masa maxima*/
  h_xmin[4] = 0.00f; /* ab minimo */
  h_xmax[4] = 1.00f; /* ab maximo */
  h_xmin[5] = 0.00f; /* bc minimo */
  h_xmax[5] = 1.00f; /* bc maximo */

  /*Volumen*/
  h_xmin[6] = 0.0; 
  h_xmax[6] = 1.0;
  h_xmin[7] = 0.0;
  h_xmax[7] = 1.0;
  h_xmin[8] = 0.0;
  h_xmax[8] = 1.0;

  /*Orientacion del halo vecino*/
  h_xmin[9]  = -M_PI;
  h_xmax[9]  =  M_PI;
  h_xmin[10] =  -1.0;
  h_xmax[10] =   1.0;
  h_xmin[11] = -M_PI;
  h_xmax[11] =  M_PI;

  //h_xmin[9]  = 0.0;
  //h_xmax[9]  = 1.0;
  //h_xmin[10] = 0.0;
  //h_xmax[10] = 1.0;
  //h_xmin[11] = 0.0;
  //h_xmax[11] = 1.0;
#endif

  h_xmin[NDIM-2] = cos((float)ANGULO*M_PI/180.0);
  h_xmax[NDIM-2] = 1.0f;
  h_xmin[NDIM-1] = 0.0f;
  h_xmax[NDIM-1] = 2.0f*M_PI;

#ifdef MERCHAN
  float norma_merchan = integra_merchan(dimGrid,dimBlock,devStates);
#endif
#ifdef DOSH
  float norma_align = integra_align(dimGrid,dimBlock,devStates);
#endif

  /*Copia los limites de integracion al device*/
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_xmin),h_xmin,NDIM*sizeof(float)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_xmax),h_xmax,NDIM*sizeof(float)));

  /*Calcula el hipervolumen de integracion*/
  volumen = 1.0f;
  for(i = 0; i < NDIM-2; i++){
    volumen *= (h_xmax[i] - h_xmin[i]);
  }

  /*Calcula la memoria total en el device*/
  size_t memfree, memtot;
  HANDLE_ERROR(hipMemGetInfo(&memfree,&memtot));
  printf("Memoria CUDA Total: %8.3lf Mb Used: %8.3lf Mb free: %8.3lf Mb \n",
         (float)memtot/1024.0f/1024.0f,(float)(memtot-memfree)/1024.0f/1024.0f,
         (float)memfree/1024.0f/1024.0f);

  /*Imprime alguna informacion*/
  printf("--------------------------\n");
  printf("  Volumen:  %E\n",volumen);
  printf("  RHOMEDIO: %E\n",RHOMEDIO);
  printf("  NCMEDIO:  %E\n",ncmedio);
#ifdef CG
  printf("  NGMEDIO: %E\n",ngmedio);
#endif
  printf("  NormaForma: %E\n",norma_merchan);
#ifdef DOSH
  printf("  NormaAlign: %E\n",norma_align);
#endif
  printf("  RNGs:  %d\n",RNGS);
  printf("  LAZOS: %d\n",LAZOS);
  printf("--------------------------\n");

  printf("Computando termino de %s....\n",term);
  fflush(stdout);

  //float dpaso;
  //dpaso = (PASOMAX - PASOMIN)/(float)NPASOS;

  /*Recorre las 3 direcciones j=0(x),1(y),2(z)*/
  for(j = 0; j < 3; j++){
    /*Abre archivo de salida*/
    sprintf(filename,"funcorr_%1d_%s.%02d",j,term,run);
    pfout = fopen(filename,"w");

    /*En cada direccion hace NPASOS pasos*/
		int ntry = 0;
    float limite = 0.01;
    for(i = 0; i < NPASOS; i++){
      /*Setea posicion en la direccion dada*/
//#ifdef DOSH
//      h_radio = dpaso*(float)(i) + PASOMIN;
//      h_radio = powf(10.0f,h_radio);
//#else
      h_radio = h_radio_vector[i];
//#endif

      /*Lanza kernel*/
      integra<<<dimGrid,dimBlock>>>(devStates,h_radio,j);
      hipDeviceSynchronize();
      /*Termina kernel*/

      CHECK_KERNEL_SUCCESS();

      /*Copia sumatorias al host y termina de reducir en el host*/
      HANDLE_ERROR(hipMemcpyFromSymbol(h_int,HIP_SYMBOL(d_int),(RNGS/THREADS_PER_BLOCK)*sizeof(float)));
      HANDLE_ERROR(hipMemcpyFromSymbol(h_sig,HIP_SYMBOL(d_sig),(RNGS/THREADS_PER_BLOCK)*sizeof(float)));

      /*Termina de reducir en el host*/
      r = 0.0; s = 0.0;
      for(l = 0; l < RNGS/THREADS_PER_BLOCK; l++){
        r += h_int[l];
        s += h_sig[l];
      }

      /*Estima la integral y el sigma*/
      r /= (float)((long)RNGS*(long)LAZOS);
      s /= (float)((long)RNGS*(long)LAZOS);
      s -= (r*r);
      s /= (float)((long)RNGS*(long)LAZOS);
      s  = sqrt(s);

      s *= volumen;
      r *= volumen;

      r /= ncmedio;
      s /= ncmedio;

      r /= norma_merchan;
      s /= norma_merchan;

#ifdef DOSH
      r /= norma_align;
      s /= norma_align;
      r /= norma_merchan;
      s /= norma_merchan;
#endif

#ifdef CG
      r /= ngmedio;
      s /= ngmedio;
#endif

			if((s/r) > limite){
				i--;
				ntry++;
				if(ntry > 10){
					limite += 0.005;
					ntry = 0;
				}
			} else {
        /*Imprime en file de salida*/
        fprintf(pfout,"%e %e %e\n",h_radio,r,s);
			}
    }
    /*Cierra archivo de salida*/
    fclose(pfout);
  }

  /*Libera memoria allocateada en el device*/
  HANDLE_ERROR(hipFree(devStates));

  /*Computa el tiempo total utilizado en el device*/
  chrono(STOP,&time);
  elapsed += time;
  sprintf(message,"Tiempo: %lf [seg] \n", elapsed);RED(message);

  /*Fin del programa*/
  return(EXIT_SUCCESS);
}
