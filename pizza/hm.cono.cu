#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include <assert.h>
#include <omp.h>
#include "constantes.h"
#include "colores.h"
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include "HandleError.cu"

#define SIGMA   0.50000f
#define SIGMA2  0.25000f
#define SIGMA3  0.12500f

#ifndef ANGULO
#define ANGULO 45
#endif

#define TRACERS_MASA_MIN 9.0f
#define TRACERS_MASA_MAX 15.0f

/*Cantidad total de hilos (RNG) que se van a tirar*/
#define RNGS 65536

/*Cantidad de veces que se lanza el Kernel de integracion*/
#define LAZOS 500
#define LAZOSPLUS 10000

/*Cantidad de Threads por Block*/
#ifndef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK 128
#endif

/*Tarjeta sobre la cual corre*/
#ifndef DEVICE
#define DEVICE 0
#endif

/*Intervalo espacial para la correlacion lineal*/
#define RANGO 10.0f

/*Grosor de la pizza*/
#ifdef DOSH
#define RANGOZ 50.0f
#else
#define RANGOZ 5.0f
#endif

/*Numero de dimensiones de la integral*/
#ifdef DOSH
#define NDIM 15
#else
#define NDIM 7
#endif

/*Cantidad de pasos en cada direccion*/
#define NPASOS   50
#define PASOMIN -1.0
#define PASOMAX  2.0

#define NDIR 2

/*Vectores integral y sigma, host version*/
float h_int[RNGS/THREADS_PER_BLOCK];
float h_sig[RNGS/THREADS_PER_BLOCK];

/*Vectores integral y sigma, device version*/
__device__ float d_int[RNGS/THREADS_PER_BLOCK];
__device__ float d_sig[RNGS/THREADS_PER_BLOCK];
__device__ float d_integral[1],d_sigma[1];

/*Coeficientes de la forma y normalizacion, host version*/
float h_bc[4][3];
float h_ab[4][3];
float h_norm[4];

/*Coeficientes de la forma y normalizacion, device version*/
__device__ float d_bc[4][3];
__device__ float d_ab[4][3];
__device__ float d_norm[4];

/*Coeficientes del alineamiento, host version*/
float h_alig[10][30][5];
float h_alig_norm[10][30];
float h_alig_m[11];
float h_alig_rmin;
float h_alig_rmax;
float h_alig_dr;

/*Coeficientes del alineamiento, device version*/
__device__ float d_alig[10][30][5];
__device__ float d_alig_norm[10][30];
__device__ float d_alig_m[11];
__device__ float d_alig_rmin;
__device__ float d_alig_rmax;
__device__ float d_alig_dr;

/*Vectores de limites de las integrales, host version*/
float h_xmin[NDIM];
float h_xmax[NDIM];
float norma_funmasa;

/*Vectores de limites de las integrales, device version*/
__constant__ float d_xmin[NDIM];
__constant__ float d_xmax[NDIM];

#include "lecturas.cu"

#include "chrono.c"

/*Incluye archivo con todas las funciones necesarias*/
#include "common_functions.cu"
#include "funciones.cu"
#include "interpolacion.cu"

/*Kernel: toma un punto aleatorio en el espacio N-Dimensional
          y evalua la funcion integrando (T1h, T2h) en dicho punto.
          La evaluacion se guarda en d_int y el cuadrado en d_sig.
          Al final guarda el estado del RNG en state*/
__global__ void integra(hiprandState *state, float r, int eje){
  const unsigned int tid = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;

  /*Estado del RNG*/
  hiprandState seed = state[tid];

  const unsigned int it = threadIdx.x;
  int   i,j;
  float x[NDIM];
  float p[3];
  double value, sigma, tmp;

  __shared__ float xmin[NDIM], dx[NDIM];
  __shared__ float s_value[THREADS_PER_BLOCK];
  __shared__ float s_sigma[THREADS_PER_BLOCK];

  /*Inicializan variables*/
  if(it < NDIM){
    xmin[it] = d_xmin[it];
      dx[it] = d_xmax[it] - xmin[it];
  }

  /*Esperan hasta que todos terminen*/
  __syncthreads();

  /*Tira un numero random x de dimension NDIM*/
  value = 0.0; sigma = 0.0;
  for(j = 0; j < LAZOS; j++){
    do{
      //x[NDIM-1] = dx[NDIM-1] * hiprand_uniform(&sedd) + xmin[NDIM-1];

      switch(eje){
        case 2 :
          x[NDIM-2] = hiprand_uniform(&seed)*2.0f*PI_CUDA;
          p[0] = r*cosf(x[NDIM-2]);
          p[1] = r*sinf(x[NDIM-2]);
          break;
        case 1 :
          x[NDIM-2] = dx[NDIM-2]*hiprand_uniform(&seed)+xmin[NDIM-2];
          x[NDIM-2] = x[NDIM-2]*GRAD2RAD;
          p[0] = r*cosf(x[NDIM-2]);
          p[1] = r*sinf(x[NDIM-2]);
          break;
        case 0 :
          x[NDIM-2] = dx[NDIM-2]*hiprand_uniform(&seed)+xmin[NDIM-2];
          x[NDIM-2] = x[NDIM-2]*GRAD2RAD;
          p[1] = r*cosf(x[NDIM-2]);
          p[0] = r*sinf(x[NDIM-2]);
          break;
      }

      p[2] = hiprand_normal(&seed); /*Linea de la visual*/
      //p[2] = RANGOZ*(2.0*hiprand_uniform(&seed)-1.0); /*Linea de la visual*/

#ifdef DOSH
      for(i = 0; i <= 9; i++)
        x[i] = dx[i] * hiprand_uniform(&seed) + xmin[i];

      x[10] = hiprand_normal(&seed);
      x[11] = hiprand_normal(&seed);
      x[12] = hiprand_normal(&seed);

      tmp = x[10]*x[10] + x[11]*x[11] + x[12]*x[12];
      if(tmp < 1.E-4){
        tmp = 1.0E-2/sqrt(tmp);
        x[10] *= tmp;
        x[11] *= tmp;
        x[12] *= tmp;
        tmp = 1.0E-4;
      }

      /*sqrt(2·pi)^3 sigma^3 / exp(-tmp/2)*/
      tmp  = SQRT_TWOPI_CUBO_CUDA*SIGMA3*exp(tmp*0.5f);
      x[10] *= SIGMA;
      x[11] *= SIGMA;
      x[12] *= SIGMA;

      /*sqrt(2·pi) RANGOZ / exp(-z*z/2)*/
      //tmp  *= 2.0f*RANGOZ;
      tmp  *= SQRT_TWOPI_CUDA*RANGOZ*exp(p[2]*p[2]*0.5f);
      p[2] *= RANGOZ;

      tmp *= T2h(p,x);

      /*
      tmp = SQRT_TWOPI_CUDA*RANGOZ*exp(p[2]*p[2]*0.5f);
      p[2] *= RANGOZ;
      tmp *= interpolador(p);
      */
#else 
      for(i = 0; i <= 4; i++)
        x[i] = dx[i] * hiprand_uniform(&seed) + xmin[i];

      //x[3] = 1.0;
      //x[4] = 0.0;

      /*sqrt(2·pi) RANGOZ / exp(-tmp/2)*/
      //tmp  = 2.0f*RANGOZ;
      tmp  = SQRT_TWOPI_CUDA*RANGOZ*exp(p[2]*p[2]*0.5f);
      p[2] *= RANGOZ;

      tmp *= T1h(p,x);

      /*
      tmp  = SQRT_TWOPI_CUDA*RANGOZ*exp(p[2]*p[2]*0.5f);
      p[2] *= RANGOZ;
      tmp *= interpolador(p);
      */
#endif

      if(isfinite(tmp))break;
    }while(1);

    value += tmp;
    sigma += tmp*tmp;
  }

  /*Guarda en la memoria compartida*/
  s_value[it] = value;
  s_sigma[it] = sigma;

  __syncthreads();

  if(it == 0){
    value = 0.0f; sigma = 0.0f;
    for(i = 0; i < THREADS_PER_BLOCK; i++){
      value += s_value[i];
      sigma += s_sigma[i];
    }
    j = blockIdx.x;
    /*Suma a la memoria global*/
    d_int[j] = value;
    d_sig[j] = sigma;
  }

  /*Guarda el estado del RNG*/
  state[tid] = seed;
}

__global__ void suma(void){
  const unsigned int tid = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
  int inext;

  inext = RNGS/THREADS_PER_BLOCK/2;
  while(inext >= 1){
    if(tid < inext){
      d_int[tid] += d_int[tid+inext];
      d_sig[tid] += d_sig[tid+inext];
    }
    inext = inext/2;
    __syncthreads();
  }

  if(tid == 0){
    d_integral[0] = d_int[0];
    d_sigma[0] = d_sig[0];
  }
}

/*Funcion que imprime las propiedades de la placa*/
__inline__ void printDevProp(hipDeviceProp_t devProp){
  printf("#############################################\n");
  printf(" Running on device:         %s\n",  devProp.name);
  printf(" Total global memory:       %zu\n", devProp.totalGlobalMem);
  printf(" Total shared memory/block: %zu\n", devProp.sharedMemPerBlock);
  printf(" Total registers/block:     %d\n",  devProp.regsPerBlock);
  printf(" Maximum threads/block:     %d\n",  devProp.maxThreadsPerBlock);
  printf(" Total constant memory:     %zu\n", devProp.totalConstMem);
  printf(" Kernel execution timeout:  %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
  printf(" WarpSize:                  %d\n",devProp.warpSize);
  printf(" Compute Capability:        %d%d\n",devProp.major,devProp.minor);
  printf("#############################################\n");
}

int setseed(void){
  int seed;
  FILE *pf;
  pf = fopen("/dev/urandom","r");
  fread(&seed,sizeof(int),1,pf);
  fclose(pf);
  return seed;
}

int main(int argc, char **argv){
  double time,elapsed;
  FILE  *pfout;
  char  filename[200],term[200];
  int   i,j,l;
  int   run;
  float r,s;
  float volumen;
  float h_radio;
  hiprandState *devStates;

  elapsed = 0.0f;
  chrono(START,&time);

  run = (argc > 1)? atoi(argv[1]) : 0;

#ifdef DOSH
  sprintf(term,"2h");
#else
  sprintf(term,"1h");
#endif

  /*Setea el device a utilizar*/
  hipSetDevice(DEVICE);

  /*Lee e imprime las propiedades del device*/
  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, DEVICE);
  printDevProp(devProp);

  /*Chequea Cantidad de Threads y de Blocks*/
  assert(THREADS_PER_BLOCK <= 1024);
  assert(RNGS%THREADS_PER_BLOCK == 0); // should be divisible by blocks

  /*Setea Cantidad de Threads y de Blocks*/
  dim3 dimBlock(THREADS_PER_BLOCK,1,1);
  dim3 dimGrid(RNGS/THREADS_PER_BLOCK,1,1);

  fprintf(stdout,"Corriendo %d Blocks con %d threads cada uno\n",RNGS/THREADS_PER_BLOCK,THREADS_PER_BLOCK);

  /*Allocatea memoria para el RNG*/
  HANDLE_ERROR(hipMalloc((void **)&devStates,RNGS*sizeof(hiprandState)));

  /*Setea la semilla*/
  int seed = setseed();

  /*Setea las semillas de los RNG en el device*/
  setup_kernel<<<dimGrid,dimBlock>>>(devStates,seed);
  hipDeviceSynchronize();

  /*lee los coeficientes de los ajustes*/
  read_coefficients();

  float Numin   = Nu_M(CENTROS_MASA_MIN);
  float Numax   = Nu_M(CENTROS_MASA_MAX);
  float ncmedio = nc_medio(Numin,Numax,dimGrid,dimBlock,devStates);
  /*FUNCION DE MASA FIT*/
  //float ncmedio = nc_medio(CENTROS_MASA_MIN,CENTROS_MASA_MAX,dimGrid,dimBlock,devStates);

#ifdef CG 
  Numin   = Nu_M(TRACERS_MASA_MIN);
  Numax   = Nu_M(TRACERS_MASA_MAX);
  float ngmedio = ng_medio(Numin,Numax,dimGrid,dimBlock,devStates);
  /*FUNCION DE MASA FIT*/
  //float ngmedio = ng_medio(TRACERS_MASA_MIN,TRACERS_MASA_MAX,dimGrid,dimBlock,devStates);
#endif

#ifndef DOSH
  /*Chequea si la integral del perfil hasta el radio virial da 1*/
  float norma_perfil = normalizacion_perfil(dimGrid,dimBlock,devStates);
#endif
  
  //prueba_ng_medio();

  /*Verifica que la integral de bias(nu)*f(nu) de 1*/
  //prueba_bias_f(dimGrid,dimBlock,devStates);

  /*Calcula la normalizacion de la funcion de masa*/
  //normalizacion_func_masa(dimGrid,dimBlock,devStates);

  proyectador_de_xilin(dimGrid,dimBlock,devStates);

  /*Setea los limites de integracion*/
  /*Halo Centro*/
  h_xmin[0] = (float)CENTROS_MASA_MIN; /*Masa minima*/
  h_xmax[0] = (float)CENTROS_MASA_MAX; /*Masa maxima*/
  /*Forma*/
  h_xmin[1] = 0.10f; /* ab minimo */
  h_xmax[1] = 1.00f; /* ab maximo */
  h_xmin[2] = 0.10f; /* bc minimo */
  h_xmax[2] = 1.00f; /* bc maximo */

  h_xmin[3] = -1.0f;     /*Orientacion del Halo Centro*/ 
  h_xmax[3] =  1.0f;      
  h_xmin[4] =  0.0f;      
  h_xmax[4] =  2.0*M_PI; 

#ifdef DOSH
  /*Halo Vecino*/
  h_xmin[5] = (float)TRACERS_MASA_MIN; /*Masa minima*/
  h_xmax[5] = (float)TRACERS_MASA_MAX; /*Masa maxima*/
  h_xmin[6] = 0.10f; /* ab minimo */
  h_xmax[6] = 1.00f; /* ab maximo */
  h_xmin[7] = 0.10f; /* bc minimo */
  h_xmax[7] = 1.00f; /* bc maximo */

  h_xmin[8] =  0.0f;
  h_xmax[8] =  0.0f;
  h_xmin[9] =  0.0f;
  h_xmax[9] =  0.0f;

  h_xmin[10] =  0.0f;
  h_xmax[10] =  0.0f;
  h_xmin[11] =  0.0f;
  h_xmax[11] =  0.0f;
  h_xmin[12] =  0.0f;
  h_xmax[12] =  0.0f;
#endif

  /** Orientacion **/
  h_xmin[NDIM-2] = 0.0f;
  h_xmax[NDIM-2] = (float)ANGULO;
  h_xmin[NDIM-1] = 0.0f;
  h_xmax[NDIM-1] = 0.0f;

#ifdef MERCHAN
  float norma_merchan = integra_merchan(dimGrid,dimBlock,devStates);
#endif

  //clock_t cuenta;
  //cuenta = clock();
  //double i1,i2,a,b;
  //a = ABMEDIO; b = 0.1;
  //i1  = (2.0*(a-1.0)*b*exp(-a*a*0.5/b/b));
  //i1 -= (sqrt(2.0*M_PI)*((a-1.0)*a+b*b)*erf((a-1.0)/sqrt(2.0)/b));
  //i1 += (sqrt(2.0*M_PI)*((a-1.0)*a+b*b)*erf(a/sqrt(2.0)/b));
  //i1 -= (2.*a*b*exp(-(a-1.0)*(a-1.0)*0.5/b/b));
  //i1 *= (-0.5*b);
  //i1 /= (sqrt(2.0*M_PI)*b);

  //a = BCMEDIO; b = 0.1;
  //i2  = (2.0*(a-1.0)*b*exp(-a*a*0.5/b/b));
  //i2 -= (sqrt(2.0*M_PI)*((a-1.0)*a+b*b)*erf((a-1.0)/sqrt(2.0)/b));
  //i2 += (sqrt(2.0*M_PI)*((a-1.0)*a+b*b)*erf(a/sqrt(2.0)/b));
  //i2 -= (2.*a*b*exp(-(a-1.0)*(a-1.0)*0.5/b/b));
  //i2 *= (-0.5*b);
  //i2 /= (sqrt(2.0*M_PI)*b);

  //float norma_merchan = i1*i2;
  //double time1 = ((double)(clock()-cuenta))/((double)CLOCKS_PER_SEC);
  //printf("  NormaForma: %E time %.15E\n",norma_merchan,time1);

#ifdef DOSH
  float norma_align = integra_align(dimGrid,dimBlock,devStates);
#endif

  /*Copia los limites de integracion al device*/
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_xmin),h_xmin,NDIM*sizeof(float)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_xmax),h_xmax,NDIM*sizeof(float)));

  /*Calcula el hipervolumen de integracion*/
  volumen = 1.0f;
#ifdef DOSH
  for(i = 0; i <= 2; i++) volumen *= (h_xmax[i] - h_xmin[i]);
  for(i = 3; i <= 7; i++) volumen *= (h_xmax[i] - h_xmin[i]);
#else
  for(i = 0; i <= 2; i++) volumen *= (h_xmax[i] - h_xmin[i]);
  for(i = 3; i <= 4; i++) volumen *= (h_xmax[i] - h_xmin[i]);
#endif

  /*Calcula la memoria total en el device*/
  size_t memfree, memtot;
  HANDLE_ERROR(hipMemGetInfo(&memfree,&memtot));
  printf("Memoria CUDA Total: %8.3lf Mb Used: %8.3lf Mb free: %8.3lf Mb \n",
         (float)memtot/1024.0f/1024.0f,(float)(memtot-memfree)/1024.0f/1024.0f,
         (float)memfree/1024.0f/1024.0f);

  /*Imprime alguna informacion*/
  printf("--------------------------\n");
  printf("  Volumen:  %E\n",volumen);
  printf("  RHOMEDIO: %E\n",RHOMEDIO);
  printf("  NCMEDIO:  %E\n",ncmedio);
#ifdef CG
  printf("  NGMEDIO: %E\n",ngmedio);
#endif
  printf("  NormaForma: %E\n",norma_merchan);
#ifdef DOSH
  printf("  NormaAlign: %E\n",norma_align);
#endif
  printf("  RNGs:  %d\n",RNGS);
  printf("  LAZOS: %d\n",LAZOS);
  printf("--------------------------\n");

  printf("Computando termino de %s....\n",term);

  float dpaso;
  dpaso = (PASOMAX - PASOMIN)/(float)NPASOS;

  /*
  //sprintf(filename,"FC_3D_13.00-14.00_19.dat");
  sprintf(filename,"MODEL_3D.13.00-14.00.19.dat");
  pfout = fopen(filename,"r");
  for(j = 0; j < N_inter; j++){
    fscanf(pfout,"%f %f\n",&host_r[j],&host_f[j]);
  }
  fclose(pfout);
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_r),host_r,N_inter*sizeof(float)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_f),host_f,N_inter*sizeof(float)));
  */

  /*Recorre las 3 direcciones j=0(parallel),1(perpendicular),2(iso)*/
  for(j = 0; j < 3; j++){
    /*Abre archivo de salida*/
    sprintf(filename,"funcorr_%1d_%s.%02d",j,term,run);
    pfout = fopen(filename,"w");

    /*En cada direccion hace NPASOS pasos*/
    for(i = 0; i < NPASOS; i++){
      /*Setea posicion en la direccion dada*/
      h_radio = dpaso*(float)(i) + PASOMIN;
      h_radio = powf(10.0f,h_radio);

      /*Lanza kernel*/
      integra<<<dimGrid,dimBlock>>>(devStates,h_radio,j);
      hipDeviceSynchronize();
      /*Termina kernel*/

      CHECK_KERNEL_SUCCESS();

      /*Copia sumatorias al host y termina de reducir en el host*/
      HANDLE_ERROR(hipMemcpyFromSymbol(h_int,HIP_SYMBOL(d_int),(RNGS/THREADS_PER_BLOCK)*sizeof(float)));
      HANDLE_ERROR(hipMemcpyFromSymbol(h_sig,HIP_SYMBOL(d_sig),(RNGS/THREADS_PER_BLOCK)*sizeof(float)));

      /*Termina de reducir en el host*/
      r = 0.0; s = 0.0;
      for(l = 0; l < RNGS/THREADS_PER_BLOCK; l++){
        if(isnan(h_int[l]))printf("%f %d %d\n",h_int[l],i,j);
        r += h_int[l];
        s += h_sig[l];
      }

      /*Estima la integral y el sigma*/
      r /= (float)((long)RNGS*(long)LAZOS);
      s /= (float)((long)RNGS*(long)LAZOS);
      s -= (r*r);
      s /= (float)((long)RNGS*(long)LAZOS);
      s  = sqrt(s);

      s *= volumen;
      r *= volumen;

      r /= ncmedio;
      s /= ncmedio;

      r /= norma_merchan;
      s /= norma_merchan;

#ifndef DOSH
      r /= norma_perfil;
      s /= norma_perfil;
#endif

#ifdef DOSH
      r /= norma_align;
      s /= norma_align;
      r /= norma_merchan;
      s /= norma_merchan;
#endif

#ifdef CG
			r /= ngmedio;
			s /= ngmedio;
#endif

      /*Imprime en file de salida*/
      fprintf(pfout,"%e %e %e\n",h_radio,r,s);
    }
    /*Cierra archivo de salida*/
    fclose(pfout);
  }

  /*Libera memoria allocateada en el device*/
  HANDLE_ERROR(hipFree(devStates));

  /*Computa el tiempo total utilizado en el device*/
  chrono(STOP,&time);
  elapsed += time;
  sprintf(message,"Tiempo: %lf [seg] \n", elapsed);RED(message);

  /*Fin del programa*/
  return(EXIT_SUCCESS);
}
