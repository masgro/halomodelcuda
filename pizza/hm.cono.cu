#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include <assert.h>
#include <omp.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#include "constantes.h"
#include "HandleError.h"

//#include "HandleError.cu"

#define SIGMA   0.50000f
#define SIGMA2  0.25000f
#define SIGMA3  0.12500f

#ifndef ANGULO
#define ANGULO 45
#endif

//#define TRACERS_MASA_MIN 11.82f
#define TRACERS_MASA_MIN 10.82f
#define TRACERS_MASA_MAX 15.0f

/*Cantidad total de hilos (RNG) que se van a tirar*/
#define RNGS 65536

/*Cantidad de veces que se lanza el Kernel de integracion*/
#define LAZOS 500
#define LAZOSPLUS 10000

/*Cantidad de Threads por Block*/
#ifndef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK 128
#endif

/*Tarjeta sobre la cual corre*/
#ifndef DEVICE
#define DEVICE 0
#endif

/*Intervalo espacial para la correlacion lineal*/
#define RANGO 10.0f
/*Grosor de la pizza*/
#define RANGOZ 5.0f

/*Numero de dimensiones de la integral*/
#ifdef DOSH
#define NDIM 15
#else
#define NDIM 7
#endif

/*Cantidad de pasos en cada direccion*/
#define NPASOS   50
#define PASOMIN -1.0
#define PASOMAX  2.0

#define NDIR 2

/*Vectores integral y sigma, host version*/
float h_int[RNGS/THREADS_PER_BLOCK];
float h_sig[RNGS/THREADS_PER_BLOCK];

/*Vectores integral y sigma, device version*/
__device__ float d_int[RNGS/THREADS_PER_BLOCK];
__device__ float d_sig[RNGS/THREADS_PER_BLOCK];
__device__ float d_integral[1],d_sigma[1];

/*Coeficientes de la forma y normalizacion, host version*/
float h_bc[4][3];
float h_ab[4][3];
float h_norm[4];

/*Coeficientes de la forma y normalizacion, device version*/
__device__ float d_bc[4][3];
__device__ float d_ab[4][3];
__device__ float d_norm[4];

/*Coeficientes del alineamiento, host version*/
float h_alig[10][30][5];
float h_alig_norm[10][30];
float h_alig_m[11];
float h_alig_rmin;
float h_alig_rmax;
float h_alig_dr;

/*Coeficientes del alineamiento, device version*/
__device__ float d_alig[10][30][5];
__device__ float d_alig_norm[10][30];
__device__ float d_alig_m[11];
__device__ float d_alig_rmin;
__device__ float d_alig_rmax;
__device__ float d_alig_dr;

/*Vectores de limites de las integrales, host version*/
float h_xmin[NDIM];
float h_xmax[NDIM];
float norma_funmasa;

/*Vectores de limites de las integrales, device version*/
__constant__ float d_xmin[NDIM];
__constant__ float d_xmax[NDIM];

#include "lecturas.cu"

#include "chrono.c"

/*Incluye archivo con todas las funciones necesarias*/
#include "common_functions.cu"
#include "funciones.cu"

/*Kernel: toma un punto aleatorio en el espacio N-Dimensional
          y evalua la funcion integrando (T1h, T2h) en dicho punto.
          La evaluacion se guarda en d_int y el cuadrado en d_sig.
          Al final guarda el estado del RNG en state*/
__global__ void integra(hiprandState *state, float r, int eje){
  const unsigned int tid = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;

  /*Estado del RNG*/
  hiprandState seed = state[tid];

  const unsigned int it = threadIdx.x;
  int   i,j;
  float x[NDIM];
  float p[3];
  double value, sigma, tmp;

  __shared__ float xmin[NDIM], dx[NDIM];
  __shared__ float s_value[THREADS_PER_BLOCK];
  __shared__ float s_sigma[THREADS_PER_BLOCK];

  /*Inicializan variables*/
  if(it < NDIM){
    xmin[it] = d_xmin[it];
      dx[it] = d_xmax[it] - xmin[it];
  }

  /*Esperan hasta que todos terminen*/
  __syncthreads();

  /*Tira un numero random x de dimension NDIM*/
  value = 0.0; sigma = 0.0;
  for(j = 0; j < LAZOS; j++){
    do{
      //x[NDIM-1] = dx[NDIM-1] * hiprand_uniform(&sedd) + xmin[NDIM-1];

      switch(eje){
        case 2 :
          x[NDIM-2] = hiprand_uniform(&seed)*2.0f*PI_CUDA;
          p[0] = r*cosf(x[NDIM-2]);
          p[1] = r*sinf(x[NDIM-2]);
          break;

        case 1 :
          x[NDIM-2] = dx[NDIM-2]*hiprand_uniform(&seed)+xmin[NDIM-2];
          x[NDIM-2] = x[NDIM-2]*GRAD2RAD;
          p[0] = r*cosf(x[NDIM-2]);
          p[1] = r*sinf(x[NDIM-2]);
          break;

        case 0 :
          x[NDIM-2] = dx[NDIM-2]*hiprand_uniform(&seed)+xmin[NDIM-2];
          x[NDIM-2] = x[NDIM-2]*GRAD2RAD;
          p[1] = r*cosf(x[NDIM-2]);
          p[0] = r*sinf(x[NDIM-2]);
          break;
      }

      p[2] = hiprand_normal(&seed); /*Linea de la visual*/
      //p[2] = 2.0f*RANGOZ*hiprand_uniform(&seed) - RANGOZ; /*Linea de la visual*/

      //if(hiprand_uniform(&seed) < 0.5)
      //  p[0] = -p[0];
      //if(hiprand_uniform(&seed) < 0.5)
      //  p[1] = -p[1];

#ifdef DOSH
      for(i = 0; i <= 9; i++)
        x[i] = dx[i] * hiprand_uniform(&seed) + xmin[i];
      
      x[10] = hiprand_normal(&seed);
      x[11] = hiprand_normal(&seed);
      x[12] = hiprand_normal(&seed);

      tmp = x[10]*x[10] + x[11]*x[11] + x[12]*x[12];
      if(tmp < 1.E-32){
        tmp = 1.0E-16/sqrt(tmp);
        x[10] *= tmp;
        x[11] *= tmp;
        x[12] *= tmp;
        tmp = 1.0E-32;
      }

      /*sqrt(2·pi)^3 sigma^3 / exp(-tmp/2)*/
      tmp  = SQRT_TWOPI_CUBO_CUDA*SIGMA3*exp(tmp*0.5f);
      x[10] *= SIGMA;
      x[11] *= SIGMA;
      x[12] *= SIGMA;

      /*sqrt(2·pi) RANGOZ / exp(-z*z/2)*/
      //tmp  *= 2.0f*RANGOZ;
      tmp  *= SQRT_TWOPI_CUDA*RANGOZ*exp(p[2]*p[2]*0.5f);
      p[2] *= RANGOZ;

      tmp *= T2h(p,x);
#else 
      for(i = 0; i <= 4; i++)
        x[i] = dx[i] * hiprand_uniform(&seed) + xmin[i];

      /*sqrt(2·pi) RANGOZ / exp(-tmp/2)*/
      //tmp  = 2.0f*RANGOZ;
      tmp  = SQRT_TWOPI_CUDA*RANGOZ*exp(p[2]*p[2]*0.5f);
      p[2] *= RANGOZ;

      tmp *= T1h(p,x);
#endif

      if(isfinite(tmp))break;
    }while(1);

    value += tmp;
    sigma += tmp*tmp;
  }

  /*Guarda en la memoria compartida*/
  s_value[it] = value;
  s_sigma[it] = sigma;

  __syncthreads();

  if(it == 0){
    value = 0.0; sigma = 0.0;
    for(i = 0; i < THREADS_PER_BLOCK; i++){
      value += s_value[i];
      sigma += s_sigma[i];
    }
    j = blockIdx.x;
    /*Suma a la memoria global*/
    d_int[j] = value;
    d_sig[j] = sigma;
  }

  /*Guarda el estado del RNG*/
  state[tid] = seed;
}

__global__ void suma(void){
  const unsigned int tid = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK;
  int inext;

  inext = RNGS/THREADS_PER_BLOCK/2;
  while(inext >= 1){
    if(tid < inext){
      d_int[tid] += d_int[tid+inext];
      d_sig[tid] += d_sig[tid+inext];
    }
    inext = inext/2;
    __syncthreads();
  }

  if(tid == 0){
    d_integral[0] = d_int[0];
    d_sigma[0] = d_sig[0];
  }
}

/*Funcion que imprime las propiedades de la placa*/
__inline__ void printDevProp(hipDeviceProp_t devProp){
  printf("#############################################\n");
  printf(" Running on device:         %s\n",  devProp.name);
  printf(" Total global memory:       %zu\n", devProp.totalGlobalMem);
  printf(" Total shared memory/block: %zu\n", devProp.sharedMemPerBlock);
  printf(" Total registers/block:     %d\n",  devProp.regsPerBlock);
  printf(" Maximum threads/block:     %d\n",  devProp.maxThreadsPerBlock);
  printf(" Total constant memory:     %zu\n", devProp.totalConstMem);
  printf(" Kernel execution timeout:  %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
  printf(" WarpSize:                  %d\n",devProp.warpSize);
  printf(" Compute Capability:        %d%d\n",devProp.major,devProp.minor);
  printf("#############################################\n");
}

int setseed(void){
  int seed;
  FILE *pf;
  pf = fopen("/dev/urandom","r");
  fread(&seed,sizeof(int),1,pf);
  fclose(pf);
  return seed;
}

int main(int argc, char **argv){
  double time,elapsed;
  double time1;
  FILE  *pfout;
  char  filename[200],term[200];
  int   i,j,l;
  int   run;
  float r,s;
  float volumen;
  float h_radio;
  //float *d_radio;
  //int   *d_eje;
  hiprandState *devStates;

  elapsed = 0.0f;
  chrono(START,&time);

  run = (argc > 1)? atoi(argv[1]) : 0;

#ifdef DOSH
  sprintf(term,"2h");
#else
  sprintf(term,"1h");
#endif

  /*Setea el device a utilizar*/
  hipSetDevice(DEVICE);

  /*Lee e imprime las propiedades del device*/
  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, DEVICE);
  printDevProp(devProp);

  //HANDLE_ERROR(hipMalloc((void **)&d_radio,sizeof(float)));
  //HANDLE_ERROR(hipMalloc((void **)&d_eje,sizeof(int)));
  
  /*Chequea Cantidad de Threads y de Blocks*/
  assert(THREADS_PER_BLOCK <= 1024);
  assert(RNGS%THREADS_PER_BLOCK == 0); // should be divisible by blocks

  /*Setea Cantidad de Threads y de Blocks*/
  dim3 dimBlock(THREADS_PER_BLOCK,1,1);
  dim3 dimGrid(RNGS/THREADS_PER_BLOCK,1,1);

  fprintf(stdout,"Corriendo %d Blocks con %d threads cada uno\n",
                        RNGS/THREADS_PER_BLOCK,THREADS_PER_BLOCK);

  /*Allocatea memoria para el RNG*/
  HANDLE_ERROR(hipMalloc((void **)&devStates,RNGS*sizeof(hiprandState)));

  /*Setea la semilla*/
  int seed = setseed();

  /*Setea las semillas de los RNG en el device*/
  setup_kernel<<<dimGrid,dimBlock>>>(devStates,seed);
  hipDeviceSynchronize();

  /*lee los coeficientes de los ajustes*/
  read_coefficients();

  float Numin   = Nu_M(CENTROS_MASA_MIN);
  float Numax   = Nu_M(CENTROS_MASA_MAX);
  float ncmedio = nc_medio(Numin,Numax,dimGrid,dimBlock,devStates);
  /*FUNCION DE MASA FIT*/
  //float ncmedio = nc_medio(CENTROS_MASA_MIN,CENTROS_MASA_MAX,dimGrid,dimBlock,devStates);

#ifdef CG 
  Numin   = Nu_M(TRACERS_MASA_MIN);
  Numax   = Nu_M(TRACERS_MASA_MAX);
  float ngmedio = ng_medio(Numin,Numax,dimGrid,dimBlock,devStates);
  /*FUNCION DE MASA FIT*/
  //float ngmedio = ng_medio(TRACERS_MASA_MIN,TRACERS_MASA_MAX,dimGrid,dimBlock,devStates);
#endif

  /*Chequea si la integral del perfil hasta el radio virial da 1*/
  float norma_perfil = normalizacion_perfil(dimGrid,dimBlock,devStates);
  
  prueba_ng_medio();

  /*Verifica que la integral de bias(nu)*f(nu) de 1*/
  //prueba_bias_f(dimGrid,dimBlock,devStates);

  /*Calcula la normalizacion de la funcion de masa*/
  //normalizacion_func_masa(dimGrid,dimBlock,devStates);

  /*Setea los limites de integracion*/
  /*Halo Centro*/
  h_xmin[0] = (float)CENTROS_MASA_MIN; /*Masa minima*/
  h_xmax[0] = (float)CENTROS_MASA_MAX; /*Masa maxima*/
  /*Forma*/
  h_xmin[1] = 0.00f; /* ab minimo */
  h_xmax[1] = 1.00f; /* ab maximo */
  h_xmin[2] = 0.00f; /* bc minimo */
  h_xmax[2] = 1.00f; /* bc maximo */

  //h_xmin[1] = ABMEDIO; /* ab minimo */
  //h_xmax[1] = ABMEDIO; /* ab maximo */
  //h_xmin[2] = BCMEDIO; /* bc minimo */
  //h_xmax[2] = BCMEDIO; /* bc maximo */

  h_xmin[3] =  0.0f;     /*Orientacion del Halo Centro*/ 
  h_xmax[3] =  1.0f;      
  h_xmin[4] =  0.0f;      
  h_xmax[4] =  0.5f*M_PI; 

#ifdef DOSH
  /*Halo Vecino*/
  h_xmin[5] = (float)TRACERS_MASA_MIN; /*Masa minima*/
  h_xmax[5] = (float)TRACERS_MASA_MAX; /*Masa maxima*/
  h_xmin[6] = 0.0f; /* ab minimo */
  h_xmax[6] = 0.0f; /* ab maximo */
  h_xmin[7] = 0.0f; /* bc minimo */
  h_xmax[7] = 0.0f; /* bc maximo */

  h_xmin[8] =  0.0f;
  h_xmax[8] =  0.0f;
  h_xmin[9] =  0.0f;
  h_xmax[9] =  0.0f;

  h_xmin[10] =  0.0f;
  h_xmax[10] =  0.0f;
  h_xmin[11] =  0.0f;
  h_xmax[11] =  0.0f;
  h_xmin[12] =  0.0f;
  h_xmax[12] =  0.0f;
#endif

  /** Orientacion **/
  h_xmin[NDIM-2] = 0.0f;
  h_xmax[NDIM-2] = (float)ANGULO;
  h_xmin[NDIM-1] = 0.0f;
  h_xmax[NDIM-1] = 0.0f;

  clock_t cuenta;
  //cuenta = clock();
  //float norma_merchan = integra_merchan(dimGrid,dimBlock,devStates);
  //time1 = ((double)(clock()-cuenta))/((double)CLOCKS_PER_SEC);
  //printf("  NormaForma: %E time %.15E\n",norma_merchan,time1);

  cuenta = clock();
  double i1,i2,a,b;
  a = ABMEDIO; b = 0.1;
  i1  = (2.0*(a-1.0)*b*exp(-a*a*0.5/b/b));
  i1 -= (sqrt(2.0*M_PI)*((a-1.0)*a+b*b)*erf((a-1.0)/sqrt(2.0)/b));
  i1 += (sqrt(2.0*M_PI)*((a-1.0)*a+b*b)*erf(a/sqrt(2.0)/b));
  i1 -= (2.*a*b*exp(-(a-1.0)*(a-1.0)*0.5/b/b));
  i1 *= (-0.5*b);
  i1 /= (sqrt(2.0*M_PI)*b);

  a = BCMEDIO; b = 0.1;
  i2  = (2.0*(a-1.0)*b*exp(-a*a*0.5/b/b));
  i2 -= (sqrt(2.0*M_PI)*((a-1.0)*a+b*b)*erf((a-1.0)/sqrt(2.0)/b));
  i2 += (sqrt(2.0*M_PI)*((a-1.0)*a+b*b)*erf(a/sqrt(2.0)/b));
  i2 -= (2.*a*b*exp(-(a-1.0)*(a-1.0)*0.5/b/b));
  i2 *= (-0.5*b);
  i2 /= (sqrt(2.0*M_PI)*b);

  float norma_merchan = i1*i2;
  time1 = ((double)(clock()-cuenta))/((double)CLOCKS_PER_SEC);
  printf("  NormaForma: %E time %.15E\n",norma_merchan,time1);
  


#ifdef DOSH
  float norma_align = integra_align(dimGrid,dimBlock,devStates);
#endif

  /*Copia los limites de integracion al device*/
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_xmin),h_xmin,NDIM*sizeof(float)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_xmax),h_xmax,NDIM*sizeof(float)));

  /*Calcula el hipervolumen de integracion*/
  volumen = 1.0f;
#ifdef DOSH
  for(i = 0; i <= 4; i++) volumen *= (h_xmax[i] - h_xmin[i]);
  //for(i = 3; i <= 4; i++) volumen *= (h_xmax[i] - h_xmin[i]);
  for(i = 5; i <= 5; i++) volumen *= (h_xmax[i] - h_xmin[i]);
#else
  for(i = 0; i <= 4; i++) volumen *= (h_xmax[i] - h_xmin[i]);
  //for(i = 3; i <= 4; i++) volumen *= (h_xmax[i] - h_xmin[i]);
#endif

  /*Calcula la memoria total en el device*/
  size_t memfree, memtot;
  HANDLE_ERROR(hipMemGetInfo(&memfree,&memtot));
  printf("Memoria CUDA Total: %8.3lf Mb Used: %8.3lf Mb free: %8.3lf Mb \n",
         (float)memtot/1024.0f/1024.0f,(float)(memtot-memfree)/1024.0f/1024.0f,
         (float)memfree/1024.0f/1024.0f);

  /*Imprime alguna informacion*/
  printf("--------------------------\n");
  printf("  Volumen:  %E\n",volumen);
  printf("  RHOMEDIO: %E\n",RHOMEDIO);
  printf("  NCMEDIO:  %E\n",ncmedio);
#ifdef CG
  printf("  NGMEDIO: %E\n",ngmedio);
#endif
  printf("  NormaForma: %E\n",norma_merchan);
#ifdef DOSH
  printf("  NormaAlign: %E\n",norma_align);
#endif
  printf("  RNGs:  %d\n",RNGS);
  printf("  LAZOS: %d\n",LAZOS);
  printf("--------------------------\n");

  printf("Computando termino de %s....\n",term);

  float dpaso;
  dpaso = (PASOMAX - PASOMIN)/(float)NPASOS;

  /*Recorre las 3 direcciones j=0(parallel),1(perpendicular),2(iso)*/
  for(j = 0; j < 3; j++){
    /*Abre archivo de salida*/
    sprintf(filename,"funcorr_%1d_%s.%02d",j,term,run);
    pfout = fopen(filename,"w");

    /*En cada direccion hace NPASOS pasos*/
    for(i = 0; i < NPASOS; i++){
      /*Setea posicion en la direccion dada*/
      h_radio = dpaso*(float)(i) + PASOMIN;
      h_radio = powf(10.0f,h_radio);

      /*Copia posicion al device*/
      //HANDLE_ERROR(hipMemcpy(d_radio,&h_radio,sizeof(float),hipMemcpyHostToDevice));
      //HANDLE_ERROR(hipMemcpy(d_eje,&j,sizeof(int),hipMemcpyHostToDevice));

      /*Lanza kernel*/
      integra<<<dimGrid,dimBlock>>>(devStates,h_radio,j);
      hipDeviceSynchronize();

      //suma<<<dimGrid,dimBlock>>>();
      //hipDeviceSynchronize();
      /*Termina kernel*/

      CHECK_KERNEL_SUCCESS();

      ////*Copia sumatorias al host y termina de reducir en el host*/
      HANDLE_ERROR(hipMemcpyFromSymbol(h_int,HIP_SYMBOL(d_int),(RNGS/THREADS_PER_BLOCK)*sizeof(float)));
      HANDLE_ERROR(hipMemcpyFromSymbol(h_sig,HIP_SYMBOL(d_sig),(RNGS/THREADS_PER_BLOCK)*sizeof(float)));

      /*Termina de reducir en el host*/
      r = 0.0; s = 0.0;
      for(l = 0; l < RNGS/THREADS_PER_BLOCK; l++){
        if(isnan(h_int[l]))printf("%f %d %d\n",h_int[l],i,j);
        r += h_int[l];
        s += h_sig[l];
      }

      //HANDLE_ERROR(hipMemcpyFromSymbol(&r,HIP_SYMBOL(d_integral),sizeof(float)));
      //HANDLE_ERROR(hipMemcpyFromSymbol(&s,HIP_SYMBOL(d_sigma),sizeof(float)));

      /*Estima la integral y el sigma*/
      r /= (float)((long)RNGS*(long)LAZOS);
      s /= (float)((long)RNGS*(long)LAZOS);
      s -= (r*r);
      s /= (float)((long)RNGS*(long)LAZOS);
      s  = sqrt(s);

      s *= volumen;
      r *= volumen;

      r /= ncmedio;
      s /= ncmedio;

      r /= norma_merchan;
      s /= norma_merchan;

#ifndef DOSH
      r /= norma_perfil;
      s /= norma_perfil;
#endif

#ifdef DOSH
      r /= norma_align;
      s /= norma_align;
      //r /= norma_merchan;
      //s /= norma_merchan;
#endif

#ifdef CG
			r /= ngmedio;
			s /= ngmedio;
#endif

      /*Imprime en file de salida*/
      fprintf(pfout,"%e %e %e\n",h_radio,r,s);
    }
    /*Cierra archivo de salida*/
    fclose(pfout);
  }

  /*Libera memoria allocateada en el device*/
  HANDLE_ERROR(hipFree(devStates));

  /*Computa el tiempo total utilizado en el device*/
  chrono(STOP,&time);
  elapsed += time;
  printf("Tiempo: %lf [seg] \n", elapsed);

  /*Fin del programa*/
  return(EXIT_SUCCESS);
}
